#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "cu.cuh"

using namespace cv;
using namespace std;


__global__ void matrix_symmetry_check_no_atomic(int* matrix, const int countLine, const int countColumn, int* vec)
{
	int iLine = blockIdx.x * blockDim.x + threadIdx.x;																		//3
	int iFirstElementLine = iLine * 1024;																					//2
	int iLastElementLine = iFirstElementLine + 511;																			//2
	if (iFirstElementLine < countLine * countColumn && iLastElementLine < countLine * countColumn && iLine < countLine)		//7
	{
		for (int i = 0; i < 512; i++)																						//3 � ����� + 1 ���
		{
			if (vec[iLine] == 1)																							//8
			{
				vec[iLine] = matrix[iFirstElementLine + i] == matrix[iLastElementLine - i] ? 1 : 0;							//25
			}
		}																													//36 * 512 + 1 = 18433
	}
}


void fillIntVecRes(int* vec, int n);

int sumElementsInRes(int* vec, int n);

int LaunchGPULab3new(int* host_matrix, int countLine, int countColumn)
{
	int* host_is_symmetrical_lines = new int[countLine];
	fillIntVecRes(host_is_symmetrical_lines, countLine);

	int* dev_matrix;
	int* dev_is_symmetrical_lines;

	const int size_matrix = countLine * countColumn * sizeof(int);
	const int size_vector = countLine * sizeof(int);

	// ���������� ������ �������
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	// ������� ������ � ��������� ���������� ����
	hipEventCreate(&start);
	hipEventCreate(&stop);

	CHECK(hipMalloc((void**)&dev_matrix, size_matrix));											// ��������� ������ �� ����������
	CHECK(hipMalloc((void**)&dev_is_symmetrical_lines, size_vector));								// ��������� ������ �� ����������

	CHECK(hipMemcpy(dev_matrix, host_matrix, size_matrix, hipMemcpyHostToDevice));								// �������� �������� �� ����������
	CHECK(hipMemcpy(dev_is_symmetrical_lines, host_is_symmetrical_lines, size_vector, hipMemcpyHostToDevice));	// �������� �������� �� ����������

	int countThreads = countColumn;																
	int countBlocks = (countLine + countThreads - 1) / countThreads;				// ���������� ������������ ������
	printf("Blocks: %i\t Threads: %i\n", countBlocks, countThreads);

	hipEventRecord(start, 0);                                                          // ����������� start � �������� �����

	matrix_symmetry_check_no_atomic KERNEL_ARGS2(countBlocks, countThreads) (dev_matrix, countLine, countColumn, dev_is_symmetrical_lines);

	hipEventRecord(stop, 0);                                                           // ����������� stop � �������� �����
	hipEventSynchronize(stop);                                                         // ���������� ��������� ��������� ���������� ����, ��������� ����������� ������������� �� ������� stop
	hipEventElapsedTime(&gpuTime, start, stop);                                        // ����������� ����� ����� ��������� start � stop
	printf("Time on GPU = %f milliseconds\n", gpuTime);                                 // ����� ����� ���������� � �������������
	hipEventDestroy(start);                                                            // ������� ������� start
	hipEventDestroy(stop);                                                             // ������� ������� stop

	CHECK(hipGetLastError());                                                          // �������� �� ������

	CHECK(hipMemcpy(host_is_symmetrical_lines, dev_is_symmetrical_lines, size_vector, hipMemcpyDeviceToHost));	// �������� �������� � ����������

	CHECK(hipFree(dev_matrix));														// �������
	CHECK(hipFree(dev_is_symmetrical_lines));											// �������

	//printf("Memory bandwidth: %f Gb/s\n", ((size_matrix / 1024.0 / 1024.0 / 1024.0) / (gpuTime / 1000.0)));
	//printf("The matrix is symmetric ( %s )\n", host_is_symmetric ? "true" : "false");
	printf("The sum of the vector elements = %i", sumElementsInRes(host_is_symmetrical_lines, countLine));

	return 0;
}