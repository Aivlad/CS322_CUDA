#include "hip/hip_runtime.h"

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "cu.cuh"

using namespace cv;
using namespace std;

// GPU: ���������� � ��������� ������ ����������� �������� "�������"
// threadIdx - ���������� ������ � ����� �������
// blockIdx - ���������� ����� ������� � �����
// blockDim - ������� ����� �������
// gridDim - ������� ����� ������ �������
__global__ void negative(uchar* img, int channel, int N)
{
    int i = 3 * (blockIdx.x * blockDim.x + threadIdx.x);
    if (i >= N)
        return;
    img[i + channel] = 255 - img[i + channel];
}

int LaunchGPU(int channel, string path)
{
    // ����� ���������� �� ���������
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // ���������� ���������� � ������ ����������
    print_cuda_device_info(prop);       // ����� ���������� �� ����������� ����������

    // ������ �����
    Mat image = imread(path);
    if (!image.data)    // �������� ����������� �����
    {
        cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    // ���������� ������ �������
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    // ������� ������ � ��������� ���������� ����
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ��������� ����������� ������ ��������� �����������
    int _width = image.cols;
    int _height = image.rows;
    int _type = image.type();

    // ������ � ������������
    uchar* host_img = image.data;                                                       // �������� ��������� �� ������
    size_t N = image.rows * image.cols * 3;                                             // ���������� N - ���������� ���� rgb ������� (� ������� ������� �� �� 3)

    uchar* dev_img;                                                                     // ���������� ���������� ��� �����������

    CHECK(hipMalloc((void**)&dev_img, N * sizeof(uchar)));                             // ��������� ������ �� ����������

    CHECK(hipMemcpy(dev_img, host_img, N * sizeof(uchar), hipMemcpyHostToDevice));    // �������� �������� �� ����������

    int countBlocks = (N / 3 + 511) / 512;                                                  // ���������� ������������ ������
    int countThreads = 512;                                                             // ���������� ����� ��� ������� �����
    printf("Blocks: %i\t Threads: %i\n", countBlocks, countThreads);

    hipEventRecord(start, 0);                                                          // ����������� start � �������� �����

    negative KERNEL_ARGS2(countBlocks, countThreads) (dev_img, channel, N);             // ������ negative() �� ���� GPU

    hipEventRecord(stop, 0);                                                           // ����������� stop � �������� �����
    hipEventSynchronize(stop);                                                         // ���������� ��������� ��������� ���������� ����, ��������� ����������� ������������� �� ������� stop
    hipEventElapsedTime(&gpuTime, start, stop);                                        // ����������� ����� ����� ��������� start � stop
    printf("Time on GPU = %f milliseconds\n", gpuTime);                                 // ����� ����� ���������� � �������������
    hipEventDestroy(start);                                                            // ������� ������� start
    hipEventDestroy(stop);                                                             // ������� ������� stop

    CHECK(hipGetLastError());                                                          // �������� �� ������

    CHECK(hipMemcpy(host_img, dev_img, N * sizeof(uchar), hipMemcpyDeviceToHost));    // �������� �������� � ����������

    Mat imageOut = Mat(_height, _width, _type, host_img);                               // ����������� ���������� uchar � Mat

    CHECK(hipFree(dev_img));                                                           // �������

    // ���������� �����������
    imwrite("out_img_gpu.jpg", imageOut);

    //// ����� �����������
    //namedWindow("Display window", WINDOW_AUTOSIZE); // �������� ���� �����������
    //imshow("Display window", imageOut);             // ������ ���� ���������� �����������
    //waitKey(0);                                     // �������� ������� ������� � ����
    
    return 0;
}