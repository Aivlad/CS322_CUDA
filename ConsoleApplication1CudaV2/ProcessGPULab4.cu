#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "cu.cuh"

#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

using namespace cv;
using namespace std;

__global__ void matrix_symmetry_check_shared(int* matrix, const int countLine, const int countColumn, int* vec)
{
	_/*_shared__ int cache[256];

	int iLine = blockIdx.x * blockDim.x + threadIdx.x;																	
	int iFirstElementLine = iLine * 1024;																					
	int iLastElementLine = iFirstElementLine + 511;	*/																		


	//__shared__ int cache[1024];
	//int iCache = threadIdx.x;
	//int iVec = blockIdx.x;
	//int iMatrix = blockIdx.x * blockDim.x + threadIdx.x;
	//cache[threadIdx.x] = matrix[iMatrix];
	//__syncthreads();

	//if (iCache < countColumn / 2)
	//{
	//	if (vec[iVec] == 1)
	//	{
	//		int iSymLine = 1024 - iCache - 1;
	//		vec[iVec] = cache[iCache] == cache[iSymLine] ? atomicAnd(vec + iVec, 1) : atomicAnd(vec + iVec, 0);
	//	}
	//}


}

/*
	__global__ void matrix_symmetry_check_no_atomic(int* matrix, const int countLine, const int countColumn, int* vec)
{
	int iLine = blockIdx.x * blockDim.x + threadIdx.x;																		//3
	int iFirstElementLine = iLine * 1024;																					//2
	int iLastElementLine = iFirstElementLine + 511;																			//2
	if (iFirstElementLine < countLine * countColumn && iLastElementLine < countLine * countColumn && iLine < countLine)		//7
	{
		for (int i = 0; i < 512; i++)																						//3 � ����� + 1 ���
		{
			if (vec[iLine] == 1)																							//8
			{
				vec[iLine] = matrix[iFirstElementLine + i] == matrix[iLastElementLine - i] ? 1 : 0;							//25
			}
		}																													//36 * 512 + 1 = 18433
	}
}
	
*/

void fillIntVecRes(int* vec, int n);

int sumElementsInRes(int* vec, int n);


int LaunchGPULab4(int* host_matrix, int countLine, int countColumn)
{
	int* host_is_symmetrical_lines = new int[countLine];
	fillIntVecRes(host_is_symmetrical_lines, countLine);

	int* dev_matrix;
	int* dev_is_symmetrical_lines;

	const int size_matrix = countLine * countColumn * sizeof(int);
	const int size_vector = countLine * sizeof(int);

	// ���������� ������ �������
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	// ������� ������ � ��������� ���������� ����
	hipEventCreate(&start);
	hipEventCreate(&stop);

	CHECK(hipMalloc((void**)&dev_matrix, size_matrix));											// ��������� ������ �� ����������
	CHECK(hipMalloc((void**)&dev_is_symmetrical_lines, size_vector));								// ��������� ������ �� ����������

	CHECK(hipMemcpy(dev_matrix, host_matrix, size_matrix, hipMemcpyHostToDevice));								// �������� �������� �� ����������
	CHECK(hipMemcpy(dev_is_symmetrical_lines, host_is_symmetrical_lines, size_vector, hipMemcpyHostToDevice));	// �������� �������� �� ����������

	int countThreads = 256;																// ���������� ����� ��� ������� ����� (!!! ���-�� ����� = ���������� �������� �������)
	int countBlocks = (countLine * countColumn + countThreads - 1) / countThreads;				// ���������� ������������ ������
	printf("Blocks: %i\t Threads: %i\n", countBlocks, countThreads);

	hipEventRecord(start, 0);                                                          // ����������� start � �������� �����

	matrix_symmetry_check_shared KERNEL_ARGS2(countBlocks, countThreads) (dev_matrix, countLine, countColumn, dev_is_symmetrical_lines);

	hipEventRecord(stop, 0);                                                           // ����������� stop � �������� �����
	hipEventSynchronize(stop);                                                         // ���������� ��������� ��������� ���������� ����, ��������� ����������� ������������� �� ������� stop
	hipEventElapsedTime(&gpuTime, start, stop);                                        // ����������� ����� ����� ��������� start � stop
	printf("Time on GPU = %f milliseconds\n", gpuTime);                                 // ����� ����� ���������� � �������������
	hipEventDestroy(start);                                                            // ������� ������� start
	hipEventDestroy(stop);                                                             // ������� ������� stop

	CHECK(hipGetLastError());                                                          // �������� �� ������

	CHECK(hipMemcpy(host_is_symmetrical_lines, dev_is_symmetrical_lines, size_vector, hipMemcpyDeviceToHost));	// �������� �������� � ����������

	CHECK(hipFree(dev_matrix));														// �������
	CHECK(hipFree(dev_is_symmetrical_lines));											// �������

	//printf("Memory bandwidth: %f Gb/s\n", ((size_matrix / 1024.0 / 1024.0 / 1024.0) / (gpuTime / 1000.0)));
	//printf("The matrix is symmetric ( %s )\n", host_is_symmetric ? "true" : "false");
	printf("The sum of the vector elements = %i", sumElementsInRes(host_is_symmetrical_lines, countLine));

	return 0;
}