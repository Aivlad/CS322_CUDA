#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "cu.cuh"

#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

using namespace cv;
using namespace std;

const int portion = 1024;

__global__ void matrix_symmetry_check_shared(int* matrix, const int countLine, const int countColumn, bool* isSymmetric)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= countLine * countColumn)
		return;

	__shared__ int cache[portion];

	cache[threadIdx.x] = matrix[i];
	
	__syncthreads();

	*isSymmetric = cache[threadIdx.x] == cache[portion - 1 - threadIdx.x] ? true : false;
}

int LaunchGPULab4(int* host_matrix, int countLine, int countColumn)
{
	bool host_is_symmetric = true;	// ������������� �������, ��� ������� ������������

	int* dev_matrix;
	bool* dev_is_symmetric;

	const int size = countLine * countColumn * sizeof(int);

	// ���������� ������ �������
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	// ������� ������ � ��������� ���������� ����
	hipEventCreate(&start);
	hipEventCreate(&stop);

	CHECK(hipMalloc((void**)&dev_matrix, size));											// ��������� ������ �� ����������
	CHECK(hipMalloc((void**)&dev_is_symmetric, sizeof(bool)));                             // ��������� ������ �� ����������

	CHECK(hipMemcpy(dev_matrix, host_matrix, size, hipMemcpyHostToDevice));						// �������� �������� �� ����������
	CHECK(hipMemcpy(dev_is_symmetric, &host_is_symmetric, sizeof(bool), hipMemcpyHostToDevice));	// �������� �������� �� ����������

	int countThreads = countColumn;																// ���������� ����� ��� ������� ����� (!!! ���-�� ����� = ���������� �������� �������)
	int countBlocks = (countLine * countColumn + countThreads - 1) / countThreads;				// ���������� ������������ ������
	printf("Blocks: %i\t Threads: %i\n", countBlocks, countThreads);

	hipEventRecord(start, 0);                                                          // ����������� start � �������� �����

	matrix_symmetry_check_shared KERNEL_ARGS2(countBlocks, countThreads) (dev_matrix, countLine, countColumn, dev_is_symmetric);

	hipEventRecord(stop, 0);                                                           // ����������� stop � �������� �����
	hipEventSynchronize(stop);                                                         // ���������� ��������� ��������� ���������� ����, ��������� ����������� ������������� �� ������� stop
	hipEventElapsedTime(&gpuTime, start, stop);                                        // ����������� ����� ����� ��������� start � stop
	printf("Time on GPU = %f milliseconds\n", gpuTime);                                 // ����� ����� ���������� � �������������
	hipEventDestroy(start);                                                            // ������� ������� start
	hipEventDestroy(stop);                                                             // ������� ������� stop

	CHECK(hipGetLastError());                                                          // �������� �� ������

	CHECK(hipMemcpy(&host_is_symmetric, dev_is_symmetric, sizeof(bool), hipMemcpyDeviceToHost));	// �������� �������� � ����������

	CHECK(hipFree(dev_matrix));														// �������
	CHECK(hipFree(dev_is_symmetric));													// �������

	printf("Memory bandwidth: %f Gb/s\n", ((size / 1024.0 / 1024.0 / 1024.0) / (gpuTime / 1000.0)));
	printf("The matrix is symmetric ( %s )\n", host_is_symmetric ? "true" : "false");

	return 0;
}